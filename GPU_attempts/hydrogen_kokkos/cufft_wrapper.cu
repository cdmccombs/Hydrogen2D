
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

extern "C" {

// Forward FFT implementation using cuFFT
void forward_fft_2d(void* data, int nx, int ny) {
    hipfftHandle plan;
    hipfftResult result = hipfftPlan2d(&plan, nx, ny, HIPFFT_Z2Z);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT Plan creation failed: %d\n", result);
        return;
    }
    
    result = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)data, (hipfftDoubleComplex*)data, HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT forward transform failed: %d\n", result);
    }
    
    hipfftDestroy(plan);
}

// Backward FFT implementation using cuFFT
void backward_fft_2d(void* data, int nx, int ny) {
    hipfftHandle plan;
    hipfftResult result = hipfftPlan2d(&plan, nx, ny, HIPFFT_Z2Z);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT Plan creation failed: %d\n", result);
        return;
    }
    
    result = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)data, (hipfftDoubleComplex*)data, HIPFFT_BACKWARD);
    if (result != HIPFFT_SUCCESS) {
        printf("CUFFT inverse transform failed: %d\n", result);
    }
    
    hipfftDestroy(plan);
}

}
